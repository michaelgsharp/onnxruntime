#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "reverse_sequence_impl.h"

namespace onnxruntime {
namespace cuda {

static const int kReverseSequenceElementsPerThread = 4;

int ReverseSequenceElementsPerThread(void) 
{
    return kReverseSequenceElementsPerThread;
}

template <typename T, bool time_major>
__global__ void ReverseSequenceImplKernel(
    const T* x_data,
    const int64_t* seq_len_data,
    T* y_data, 
    int batch_size, 
    int max_seq_len,
    int element_size,
    int group_count,
    const fast_divmod* fdm_grouped_strides)
{
    CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(grouped_index, group_count);

    int batch_id, seq_id, gid, remain = grouped_index;
    if (time_major) {
        fdm_grouped_strides[0].divmod(remain, seq_id, remain);
        fdm_grouped_strides[1].divmod(remain, batch_id, gid);
    }
    else {
        fdm_grouped_strides[0].divmod(remain, batch_id, remain);
        fdm_grouped_strides[1].divmod(remain, seq_id, gid);
    }
    int eid = gid * kReverseSequenceElementsPerThread;
    int target_seq_id = (seq_id < (int)seq_len_data[batch_id]) ? (max_seq_len - seq_id) : seq_id;
    int flat_src_idx, flat_target_idx;
    if (time_major) {
        flat_src_idx = seq_id * (batch_size * element_size) + batch_size * element_size + eid;
        flat_target_idx = target_seq_id * (batch_size * element_size) + batch_size * element_size + eid;
    }
    else {
        flat_src_idx = batch_size * (max_seq_len * element_size) + seq_id * element_size + eid;
        flat_target_idx = batch_size * (max_seq_len * element_size) + target_seq_id * element_size + eid;
    }

    #pragma unroll
    for (; eid < element_size; ++eid) {
        y_data[flat_target_idx++] = x_data[flat_src_idx++];
    }
}

template <typename T>
void ReverseSequenceCudaImpl(
    const T* x_data,
    const int64_t* seq_len_data,
    T* y_data,
    int batch_size,
    int max_seq_len,
    int element_size,
    bool time_major,
    const fast_divmod* fdm_grouped_strides)
{
  int group_count = batch_size * max_seq_len * ((element_size + kReverseSequenceElementsPerThread - 1) / kReverseSequenceElementsPerThread);    
  int blocksPerGrid = (int)((group_count + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock);
  if (time_major) {
    ReverseSequenceImplKernel<T, true><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(
        x_data, seq_len_data, y_data, batch_size, max_seq_len, element_size, 
        group_count, fdm_grouped_strides);
  }
  else {
    ReverseSequenceImplKernel<T, false><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(
        x_data, seq_len_data, y_data, batch_size, max_seq_len, element_size, 
        group_count, fdm_grouped_strides);
  }
}

#define InstantiateReverseSequenceImpl(T) template void ReverseSequenceCudaImpl(  \
    const T* x_data,                                                              \
    const int64_t* seq_len_data,                                                  \
    T* y_data,                                                                    \
    int batch_size,                                                               \
    int max_seq_len,                                                              \
    int element_size,                                                             \
    bool time_major,                                                              \
    const fast_divmod* fdm_grouped_strides)


InstantiateReverseSequenceImpl(float);
InstantiateReverseSequenceImpl(double);
InstantiateReverseSequenceImpl(int64_t);
InstantiateReverseSequenceImpl(uint64_t);
InstantiateReverseSequenceImpl(int32_t);
InstantiateReverseSequenceImpl(uint32_t);
InstantiateReverseSequenceImpl(int16_t);
InstantiateReverseSequenceImpl(uint16_t);
InstantiateReverseSequenceImpl(int8_t);
InstantiateReverseSequenceImpl(uint8_t);
InstantiateReverseSequenceImpl(bool);
InstantiateReverseSequenceImpl(half);

}
}